#include "hip/hip_runtime.h"
#include <cstdint>
#include <src/utils.h>
#include <stdio.h>

// * For FP16

__device__ half warpReduceMax(half val, int warpSize)
{
    for (int offset = warpSize / 2; offset > 0; offset /= 2)
    {
        val = __hmax(val, __shfl_down_sync(0xffffffff, val, offset));
    }
    return val;
}

__device__ half warpReduceSum(half val, int warpSize)
{
    for (int offset = warpSize / 2; offset > 0; offset /= 2)
        val = __hadd(val, __shfl_down_sync(0xffffffff, val, offset));
    return val;
}

__device__ float half_to_float(uint16_t h)
{
    float f;
    asm volatile("cvt.f32.f16 %0, %1;\n" : "=f"(f) : "h"(h));
    return f;
}

__device__ float2 half2_to_float2(uint32_t v)
{
    uint16_t lo, hi;
    asm volatile("mov.b32 {%0, %1}, %2;\n" : "=h"(lo), "=h"(hi) : "r"(v));
    return make_float2(half_to_float(lo), half_to_float(hi));
}

__device__ uint32_t float2_to_half2(float2 f)
{
    union
    {
        uint32_t u32;
        uint16_t u16[2];
    } tmp;
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 800
    asm volatile("cvt.rn.f16x2.f32 %0, %1, %2;\n" : "=r"(tmp.u32) : "f"(f.y), "f"(f.x));
#else
    asm volatile("cvt.rn.f16.f32 %0, %1;\n" : "=h"(tmp.u16[0]) : "f"(f.x));
    asm volatile("cvt.rn.f16.f32 %0, %1;\n" : "=h"(tmp.u16[1]) : "f"(f.y));
#endif
    return tmp.u32;
}

__device__ float2 rotary_embedding_coefficient(const int zid, const int rot_embed_dim, const float t_step)
{
    const float inv_freq = t_step / pow(10000.0f, zid / (float)rot_embed_dim);
    return {cos(inv_freq), sin(inv_freq)};
}

__device__ float2 rotary_embedding_transform(const float2 v, const float2 coef)
{
    float2 rot_v;
    rot_v.x = coef.x * v.x - coef.y * v.y;
    rot_v.y = coef.x * v.y + coef.y * v.x;
    return rot_v;
}

__device__ uint32_t rotary_embedding_transform(const uint32_t v, const float2 coef)
{
    float2 fv = half2_to_float2(v);
    float2 rot_fv = rotary_embedding_transform(fv, coef);
    return float2_to_half2(rot_fv);
}

// 一个线程负责八个元素的旋转编码
__device__ void apply_rotary_embedding(uint4 &q, uint4 &k, int tid, int rot_embed_dim, int t_step)
{
    if (8 * tid >= rot_embed_dim)
    {
        return;
    }
    const auto coef0 = rotary_embedding_coefficient(8 * tid, rot_embed_dim, t_step);
    q.x = rotary_embedding_transform(q.x, coef0);
    k.x = rotary_embedding_transform(k.x, coef0);
    const auto coef1 = rotary_embedding_coefficient(8 * tid + 2, rot_embed_dim, t_step);
    q.y = rotary_embedding_transform(q.y, coef1);
    k.y = rotary_embedding_transform(k.y, coef1);
    const auto coef2 = rotary_embedding_coefficient(8 * tid + 4, rot_embed_dim, t_step);
    q.z = rotary_embedding_transform(q.z, coef2);
    k.z = rotary_embedding_transform(k.z, coef2);
    const auto coef3 = rotary_embedding_coefficient(8 * tid + 6, rot_embed_dim, t_step);
    q.w = rotary_embedding_transform(q.w, coef3);
    k.w = rotary_embedding_transform(k.w, coef3);
}

// for fp16 gemv

__device__ void gemv_qk(half *mat, half *vec, half *res, int n, int k, int tidx, int n_elem_per_thread)
{
    uint4 *mat_uint4 = reinterpret_cast<uint4 *>(mat);
    uint4 *vec_uint4 = reinterpret_cast<uint4 *>(vec);
    // (uint4) mat (n/8, k/8) (uint4) vec (k/8) (uint4) res (n/8)
    // 共计算n_elem_per_thread个元素的结果，偏移量为tidx*n_elem_per_thread，确保k一定可以被8整除
    // 找到对应的矩阵行后，沿着k方向循环，每次向量化拷贝8个Vec和n_elem_per_thread*8个Mat的元素，相乘后结果存储到res，注意Mat不要超出n的范围
    if (tidx * n_elem_per_thread >= n)
    {
        return;
    }
    int n_min = tidx * n_elem_per_thread;
    int n_max = min(tidx * (n_elem_per_thread + 1), n);
#pragma unroll
    for (int i = 0; i < k / 8; i++)
    {
        uint4 vec_tmp = vec_uint4[i];
        half2 *vec_h1 = (half2 *)&vec_tmp.x;
        half2 *vec_h2 = (half2 *)&vec_tmp.y;
        half2 *vec_h3 = (half2 *)&vec_tmp.z;
        half2 *vec_h4 = (half2 *)&vec_tmp.w;
        for (int j = n_min; j < n_max; j++)
        {
            uint4 mat_tmp = mat_uint4[j * k / 8 + i];
            half2 *mat_h1 = (half2 *)&mat_tmp.x;
            half2 *mat_h2 = (half2 *)&mat_tmp.y;
            half2 *mat_h3 = (half2 *)&mat_tmp.z;
            half2 *mat_h4 = (half2 *)&mat_tmp.w;
            res[j] += mat_h1->x * vec_h1->x;
            res[j] += mat_h1->y * vec_h1->y;
            res[j] += mat_h2->x * vec_h2->x;
            res[j] += mat_h2->y * vec_h2->y;
            res[j] += mat_h3->x * vec_h3->x;
            res[j] += mat_h3->y * vec_h3->y;
            res[j] += mat_h4->x * vec_h4->x;
            res[j] += mat_h4->y * vec_h4->y;
        }
    }
}

__device__ void gemv_pv(half *rP, half *sV, half *rO, int seqLen, int headDim, int tidx, int n_elem_per_thread, bool is_first, int m1_in_formula, int m2_in_formula)
{
    // rP: (4) ...[tidx*n_elem_per_thread, (tidx+1)*n_elem_per_thread) located in P [seqLen]
    // sV: (seqLen, headDim)
    // rO: (4)

    // 非第一次的情况，先修正原先的O
    if (!is_first)
    {
        for (int i = 0; i < 4; i++)
        {
            rO[i] = rO[i] * __double2half(exp(m1_in_formula - m2_in_formula));
        }
    }
    // 每个线程每次沿H维度向量化读取sV的8个元素，读取4行，共读取32个元素进入寄存器
    // 使用Warp快速通信的方式将结果累加到rO中
    for (int u = 0; u < headDim / 8; u += 1)
    {
        half buf[8] = {__float2half(0.0f), __float2half(0.0f), __float2half(0.0f), __float2half(0.0f),
                       __float2half(0.0f), __float2half(0.0f), __float2half(0.0f), __float2half(0.0f)};
        half output[4] = {__float2half(0.0f), __float2half(0.0f), __float2half(0.0f), __float2half(0.0f)};
        for (int i = 0; i < seqLen; i += n_elem_per_thread)
        {
            for (int j = 0; j < n_elem_per_thread && i + j < seqLen; j++)
            {
                uint4 rV = reinterpret_cast<uint4 *>(sV + (i + j) * headDim)[u];
                half2 *rV_h1 = (half2 *)&rV.x;
                half2 *rV_h2 = (half2 *)&rV.y;
                half2 *rV_h3 = (half2 *)&rV.z;
                half2 *rV_h4 = (half2 *)&rV.w;
                buf[0] += rP[j] * rV_h1->x;
                buf[1] += rP[j] * rV_h1->y;
                buf[2] += rP[j] * rV_h2->x;
                buf[3] += rP[j] * rV_h2->y;
                buf[4] += rP[j] * rV_h3->x;
                buf[5] += rP[j] * rV_h3->y;
                buf[6] += rP[j] * rV_h4->x;
                buf[7] += rP[j] * rV_h4->y;
            }
        }
        // 规约当前warp上的buf，广播到每个线程中，如果是当前线程负责的元素，则相加到寄存器rO上
        for (int i = 0; i < 8; ++i)
        {
            buf[i] = warpReduceSum(buf[i], 32);
        }
        if (tidx >= u * 8 && tidx < (u + 1) * 8)
        {
            if (tidx % 2 == 0)
            {
                rO[0] += buf[0];
                rO[1] += buf[1];
                rO[2] += buf[2];
                rO[3] += buf[3];
            }
            else
            {
                rO[0] += buf[4];
                rO[1] += buf[5];
                rO[2] += buf[6];
                rO[3] += buf[7];
            }
        }
    }
}

__device__ void clear_shm(half *p, const int n)
{
    for (int i = threadIdx.x; i < n; i += blockDim.x)
    {
        p[i] = 0;
    }
}

__device__ void clear_reg(half *p, const int n)
{
    for (int i = 0; i < n; i++)
    {
        p[i] = 0;
    }
}

__device__ half __max(half a, half b) {
    return a > b ? a : b;
}

__device__ half __exp(half x) {
    return hexp(x);
}

__global__ void init_half_array_kernel(half *array, half value, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        array[idx] = value;
    }
}

void init_half_array(half *array, half value, int n, int numBlocks, int blockSize) {
    init_half_array_kernel<<<numBlocks, blockSize>>>(array, value, n);
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("Init CUDA kernel error: %s\n", hipGetErrorString(error));
    }
}